#include "hip/hip_runtime.h"
#include <moderngpu/context.hxx>
#include <moderngpu/kernel_intervalmove.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/memory.hxx>
using namespace mgpu;

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <queue>
#include <vector>
using namespace std;

#include "bfs-mgpu.cuh"

namespace bfs_mgpu {

__global__ void UpdateDistanceAndVisitedKernel(const int* __restrict__ frontier,
                                               int frontier_size, int d,
                                               int* distance, int* visited) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < frontier_size; i += step) {
        distance[frontier[i]] = d;
        atomicOr(visited + (frontier[i] >> 5), 1 << (frontier[i] & 31));
    }
}

__global__ void CalculateFrontierStartsAndDegreesKernel(
    const int* __restrict__ nodes, const int* __restrict__ frontier, int n,
    int* node_frontier_starts, int* node_frontier_degrees) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < n; i += step) {
        node_frontier_starts[i] = nodes[frontier[i]];
        node_frontier_degrees[i] = nodes[frontier[i] + 1] - nodes[frontier[i]];
    }
}

__global__ void AdvanceFrontierPhase1Kernel(
    const int* __restrict__ edge_frontier, int edge_frontier_size,
    const int* __restrict__ visited, int* parent, int* edge_frontier_success) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < edge_frontier_size; i += step) {
        int v = edge_frontier[i];
        int success =
            (((visited[v >> 5] >> (v & 31)) & 1) == 0 && parent[v] == -1) ? 1
                                                                          : 0;
        if (success) parent[edge_frontier[i]] = i;
        edge_frontier_success[i] = success;
    }
}

__global__ void AdvanceFrontierPhase2Kernel(
    const int* __restrict__ edge_frontier, int edge_frontier_size,
    const int* __restrict__ parent, int* edge_frontier_success) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < edge_frontier_size; i += step)
        if (edge_frontier_success[i] && parent[edge_frontier[i]] != i)
            edge_frontier_success[i] = 0;
}

void getMemInfo() {
    size_t fr, tot;
    hipMemGetInfo(&fr, &tot);
    cout << fr / 1e6 << " / " << tot / 1e6 << endl;
}

void ParallelBFS(int n, int m, mem_t<int>& nodes, mem_t<int>& edges, int source,
                 mem_t<int>& distance, context_t& context) {
    ParallelBFS(n, m, nodes.data(), edges.data(), source, distance.data(), context);
}

void ParallelBFS(int n, int m, int const* nodes, int const* edges, int source,
                 int * distance, context_t& context) {
    mem_t<int> visited = mgpu::fill<int>(0, (n + 31) / 32, context);
    mem_t<int> parent = mgpu::fill<int>(-1, n, context);
    mem_t<int> node_frontier(n, context);
    mem_t<int> node_frontier_starts(n, context);
    mem_t<int> node_frontier_degrees(n, context);
    mem_t<int> edge_frontier(m, context);
    mem_t<int> edge_frontier_success(m, context);

    htod(node_frontier.data(), &source, 1);

    vector<int> tmp_subarray;

    // getMemInfo();

    int node_frontier_size = 1;
    int edge_frontier_size = 0;
    for (int d = 0; node_frontier_size > 0; ++d) {
        UpdateDistanceAndVisitedKernel<<<128, 128, 0, context.stream()>>>(
            node_frontier.data(), node_frontier_size, d, distance,
            visited.data());
        CalculateFrontierStartsAndDegreesKernel<<<128, 128, 0,
                                                  context.stream()>>>(
            nodes, node_frontier.data(), node_frontier_size,
            node_frontier_starts.data(), node_frontier_degrees.data());

        // hacking a bit
        dtoh(tmp_subarray,
             node_frontier_degrees.data() + node_frontier_size - 1, 1);
        edge_frontier_size = tmp_subarray.front();

        scan<scan_type_exc>(node_frontier_degrees.data(), node_frontier_size,
                            node_frontier_degrees.data(), context);

        dtoh(tmp_subarray,
             node_frontier_degrees.data() + node_frontier_size - 1, 1);
        edge_frontier_size += tmp_subarray.front();

        interval_gather(edges, edge_frontier_size,
                        node_frontier_degrees.data(), node_frontier_size,
                        node_frontier_starts.data(), edge_frontier.data(),
                        context);
        AdvanceFrontierPhase1Kernel<<<128, 128, 0, context.stream()>>>(
            edge_frontier.data(), edge_frontier_size, visited.data(),
            parent.data(), edge_frontier_success.data());
        AdvanceFrontierPhase2Kernel<<<128, 128, 0, context.stream()>>>(
            edge_frontier.data(), edge_frontier_size, parent.data(),
            edge_frontier_success.data());

        // hacking again
        dtoh(tmp_subarray,
             edge_frontier_success.data() + edge_frontier_size - 1, 1);
        node_frontier_size = tmp_subarray.front();

        scan<scan_type_exc>(edge_frontier_success.data(), edge_frontier_size,
                            edge_frontier_success.data(), context);

        dtoh(tmp_subarray,
             edge_frontier_success.data() + edge_frontier_size - 1, 1);
        node_frontier_size += tmp_subarray.front();

        interval_expand(edge_frontier.data(), node_frontier_size,
                        edge_frontier_success.data(), edge_frontier_size,
                        node_frontier.data(), context);
    }
}

// typedef unsigned long long uint64_t;

uint64_t CalculateChecksum(const vector<int>& distance) {
    uint64_t checksum = 0;
    for (int i = 0; i < distance.size(); ++i)
        if (distance[i] != -1) checksum += (uint64_t)i * (uint64_t)distance[i];
    return checksum;
}

uint64_t Time() {
    timespec tp;
    clock_gettime(CLOCK_MONOTONIC_RAW, &tp);
    return (tp.tv_nsec + (uint64_t)1000000000 * tp.tv_sec) / 1000000;
}

uint64_t ParallelBFS(const vector<int>& nodes, const vector<int>& edges,
                     int source) {
    standard_context_t context(false);
    mem_t<int> dev_nodes = to_mem(nodes, context);
    mem_t<int> dev_edges = to_mem(edges, context);
    mem_t<int> dev_distance = mgpu::fill<int>(-1, nodes.size() - 1, context);
    uint64_t t = Time();
    ParallelBFS(nodes.size() - 1, edges.size(), dev_nodes, dev_edges, source,
                dev_distance, context);
    t = Time() - t;
    cerr << "GPU: " << t << " ms" << endl;
    vector<int> distance = from_mem(dev_distance);
    return CalculateChecksum(distance);
}

uint64_t SequentialBFS(const vector<int>& nodes, const vector<int>& edges,
                       int source) {
    vector<int> distance(nodes.size() - 1, -1);
    uint64_t t = Time();
    distance[source] = 0;
    queue<int> q;
    q.push(source);
    while (!q.empty()) {
        int u = q.front();
        q.pop();
        for (int i = nodes[u]; i < nodes[u + 1]; ++i) {
            int v = edges[i];
            if (distance[v] == -1) {
                distance[v] = distance[u] + 1;
                q.push(v);
            }
        }
    }
    t = Time() - t;
    cerr << "CPU: " << t << " ms" << endl;
    return CalculateChecksum(distance);
}

}  // namespace bfs_mgpu
